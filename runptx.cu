#include <hip/hip_runtime.h>
#include <iostream>

// PTX kernel as a string (or load from file)
const char* ptx_source = "                              \n \
.version 7.0                                            \n \
.target sm_50                                           \n \
.address_size 64                                        \n \
.visible .entry vectorAdd(                              \n \
    .param .u64 vectorAdd_param_0,                      \n \
    .param .u64 vectorAdd_param_1,                      \n \
    .param .u64 vectorAdd_param_2                       \n \
) {                                                     \n \
    .reg .f32   %f<4>;                                  \n \
    .reg .b32   %r<5>;                                  \n \
    .reg .b64   %rd<11>;                                \n \
    ld.param.u64    %rd1, [vectorAdd_param_0];          \n \
    ld.param.u64    %rd2, [vectorAdd_param_1];          \n \
    ld.param.u64    %rd3, [vectorAdd_param_2];          \n \
    cvta.to.global.u64      %rd4, %rd3;                 \n \
    cvta.to.global.u64      %rd5, %rd2;                 \n \
    cvta.to.global.u64      %rd6, %rd1;                 \n \
    mov.u32         %r1, %ctaid.x;                      \n \
    mov.u32         %r2, %ntid.x;                       \n \
    mov.u32         %r3, %tid.x;                        \n \
    mad.lo.s32      %r4, %r2, %r1, %r3;                 \n \
    mul.wide.u32    %rd7, %r4, 4;                       \n \
    add.s64         %rd8, %rd6, %rd7;                   \n \
    ld.global.f32   %f1, [%rd8];                        \n \
    add.s64         %rd9, %rd5, %rd7;                   \n \
    ld.global.f32   %f2, [%rd9];                        \n \
    add.f32         %f3, %f1, %f2;                      \n \
    add.s64         %rd10, %rd4, %rd7;                  \n \
    st.global.f32   [%rd10], %f3;                       \n \
    ret;                                                \n \
}";

int main() {
    hipDevice_t cuDevice;
    hipCtx_t cuContext;
    hipModule_t cuModule;
    hipFunction_t vectorAddKernel;

    hipError_t        cuerr = hipError_t::hipSuccess;
    const char*     cumsg = nullptr;

    // Initialize CUDA driver API
    cuerr = hipInit(0);
    if (cuerr) {
        hipDrvGetErrorString(cuerr, &cumsg);
        std::cout << "cuda error! " << cumsg << " @ line:" << __LINE__ << std::endl;
        return 1;
    }

    // Get GPU device
    cuerr = hipDeviceGet(&cuDevice, 0);
    if (cuerr) {
        hipDrvGetErrorString(cuerr, &cumsg);
        std::cout << "cuda error! " << cumsg << " @ line:" << __LINE__ << std::endl;
        return 1;
    }

    // Create context
    cuerr = hipCtxCreate(&cuContext, 0, cuDevice);
    if (cuerr) {
        hipDrvGetErrorString(cuerr, &cumsg);
        std::cout << "cuda error! " << cumsg << " @ line:" << __LINE__ << std::endl;
        return 1;
    }

    // Load PTX module
    cuerr = hipModuleLoadData(&cuModule, ptx_source);
    if (cuerr) {
        hipDrvGetErrorString(cuerr, &cumsg);
        std::cout << "cuda error! " << cumsg << " @ line:" << __LINE__ << std::endl;
        return 1;
    }

    // Get the kernel function from PTX
    cuerr = hipModuleGetFunction(&vectorAddKernel, cuModule, "vectorAdd");
    if (cuerr) {
        hipDrvGetErrorString(cuerr, &cumsg);
        std::cout << "cuda error! " << cumsg << " @ line:" << __LINE__ << std::endl;
        return 1;
    }

    // Define and allocate memory for input/output arrays
    const int N = 1024;
    int h_A[N], h_B[N], h_C[N];
    int* d_A; int* d_B; int* d_C;
    hipMalloc(&d_A, N * sizeof(int));
    hipMalloc(&d_B, N * sizeof(int));
    hipMalloc(&d_C, N * sizeof(int));

    // Fill input arrays with sample data
    for (int i = 0; i < N; i++) {
        h_A[i] = i + 1;
        h_B[i] = i * 2 + 1;
    }

    // Copy data to device
    hipMemcpy(d_A, h_A, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * sizeof(int), hipMemcpyHostToDevice);

    // Set kernel parameters
    void* args[] = { &d_A, &d_B, &d_C, (void*) &N };

    // Launch the PTX kernel
    cuerr = hipModuleLaunchKernel(vectorAddKernel,
        N / 256, 1, 1,   // Grid size (blocks)
        256, 1, 1,       // Block size (threads per block)
        0, nullptr,      // Shared memory size, stream
        args, nullptr);  // Kernel arguments
    if (cuerr) {
        hipDrvGetErrorString(cuerr, &cumsg);
        std::cout << "cuda error! " << cumsg << " @ line:" << __LINE__ << std::endl;
        return 1;
    }

    // Copy result back to host
    hipMemcpy(h_C, d_C, N * sizeof(int), hipMemcpyDeviceToHost);

    // Verify the result
    for (int i = 0; i < N; i++) {
        if (h_C[i] != h_A[i] + h_B[i]) {
            std::cout << i << "\t: " << h_C[i] << " != " << h_A[i] + h_B[i] << std::endl;
        } else {
            std::cout << i << "\t: OK" << std::endl;
        }
    }

    // Clean up
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    hipModuleUnload(cuModule);
    hipCtxDestroy(cuContext);

    std::cout << "Completed successfully!" << std::endl;
    return 0;
}
